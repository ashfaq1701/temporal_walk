#include "hip/hip_runtime.h"
#include "ExponentialIndexRandomPicker.cuh"
#include <random>
#include <cmath>
#include <iostream>
#include <stdexcept>

ExponentialIndexRandomPicker::ExponentialIndexRandomPicker(bool use_gpu): use_gpu(use_gpu) {}

// Derivation available in derivations folder
int ExponentialIndexRandomPicker::pick_random(const int start, const int end, const bool prioritize_end) {
    if (start >= end) {
        throw std::invalid_argument("Start must be less than end.");
    }

    const int len_seq = end - start;

    // Generate uniform random number between 0 and 1
    const double u = generate_random_value(0.0, 1.0);

    double k;
    if (len_seq < 710) {
        // Inverse CDF formula,
        // k = ln(1 + u * (e^len seq − 1)) − 1
        k = log1p(u * expm1(len_seq)) - 1;
    } else {
        // Inverse CDF approximation for large len_seq,
        // k = len_seq + ln(u) − 1
        k = len_seq + std::log(u) - 1;
    }

    // Due to rounding, the trailing "-1" in the inverse CDF formula causes error.
    // To compensate for this we add 1 with k.
    // And bound the results within limits.
    const int rounded_index = std::max(0, std::min(static_cast<int>(k + 1), len_seq - 1));

    if (prioritize_end) {
        return start + rounded_index;
    } else {
        return start + (len_seq - 1 - rounded_index);
    }
}
